#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <string>
#include <vector>
#include <complex>
using namespace std;


#include <hip/hip_runtime.h>
//#include <hip/hip_runtime_api.h>
#include <hip/hip_complex.h>

/*
 * Do einsum('lmn,lmo->lno', x, x.conj()) / M  in series
 */
complex<float>* doProblemInSeries(complex<float>* input, int L, int M, int N) {
	complex<float>* matrix = new complex<float>[L*N*N];

	for (int z = 0;z < L;z++) {
		for (int y = 0;y < N;y++) {
			for (int x = 0;x < N;x++) {
				complex<float> t = (0.0f, 0.0f);
				for (int i = 0;i < M;i++) {
					t += input[z*M*N + i * N + x] * conj(input[z*M*N + y + i * N]);
				}
				t /= M;
				matrix[z*N*N + y * N + x] = t;
				cout << t;
			}
			cout << endl;
		}
		cout << endl;
	}
	return matrix;
}

__global__ void doProblemKernel(hipComplex *in, hipComplex *out, int L, int M, int N) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < L*N*N) {
		int z = index / (N*N);
		int y = (index - (z*N*N)) / N;
		int x = index - ((y*N) + (z*N*N));
		hipComplex t = make_hipComplex(0, 0);
		for (int i = 0;i < M;i++) {
			t = hipCaddf(t, hipCmulf(in[z*M*N + i * N + x], hipConjf(in[z*M*N + y + i * N])));
		}
		t = hipCdivf(t, make_hipComplex(M, 0));
		out[index] = t;
		//out[index] = make_hipComplex(index,x);
	}
}

void cudaInfo() {
	int dev_count;
	hipGetDeviceCount(&dev_count);
	hipDeviceProp_t dev_prop;
	for (int i = 0; i < dev_count; i++) {
		hipGetDeviceProperties(&dev_prop, i);
		cout << "max threads per block : " << dev_prop.maxThreadsPerBlock << endl;
		cout << "max block x dim : " << dev_prop.maxThreadsDim[0] << endl;
		cout << "max block y dim : " << dev_prop.maxThreadsDim[1] << endl;
		cout << "max block z dim : " << dev_prop.maxThreadsDim[2] << endl;
		cout << "max grid x dim : " << dev_prop.maxGridSize[0] << endl;
		cout << "max grid y dim : " << dev_prop.maxGridSize[1] << endl;
		cout << "max grid z dim : " << dev_prop.maxGridSize[2] << endl;
		cout << "warp size : " << dev_prop.warpSize << endl;
	}

	cout << "Done\n";
}

void loadMatrix(string filename, complex<float>* matrix, int* L, int* M, int* N) {

}

int main()
{
	cudaInfo();
	int L, M, N;

	ifstream infile; string line;
	infile.open("dat\\matrix-med.txt");
	infile >> L; infile >> M; infile >> N;
	complex<float>* matrix = new complex<float>[L*M*N];

	cout << "L,M,N:" << L << "," << M << "," << N << ",\n"
		<< "Input size " << L*M*N << " Out Size " << L*N*N
		<< endl;

	for (int l = 0; l < L; l++) {
		for (int m = 0; m < M; m++) {
			for (int n = 0; n < N; n++) {
				//infile >> matrix[l][m][n];
				infile >> matrix[l*M*N + m * N + n];
			}
		}
	}

	infile.close();


	//for (int l = 0; l < L; l++) {
	//	for (int m = 0; m < M; m++) {
	//		for (int n = 0; n < N; n++) {
	//			//cout << matrix[l][m][n];
	//			cout << matrix[l*M*N + m * N + n];
	//		}
	//		cout << endl;
	//	}
	//	cout << "\n\n";
	//}

	complex<float>* seriesResult = doProblemInSeries(matrix, L, M, N);

	for (int l = 0; l < L; l++) {
		for (int m = 0; m < N; m++) {
			for (int n = 0; n < N; n++) {
				//cout << matrix[l][m][n];
				cout << seriesResult[l*N*N + m * N + n];
			}
			cout << endl;
		}
		cout << "\n\n";
	}

	ofstream outfile;
	outfile.open("dat\\seriesoutput.txt", 'w');
	for (int l = 0; l < L; l++) {
		for (int m = 0; m < N; m++) {
			for (int n = 0; n < N; n++) {
				//cout << matrix[l][m][n];
				outfile << seriesResult[l*N*N + m * N + n];
			}
			outfile << endl;
		}
		outfile << endl;
	}
	outfile.close();

	hipError_t err = hipSuccess;
	hipComplex* h_input = (hipComplex*)matrix;
	hipComplex* d_input = NULL;
	hipMalloc((void**)&d_input, L*M*N * sizeof(hipComplex));
	hipComplex* d_output = NULL;
	hipMalloc((void**)&d_output, L*N*N * sizeof(hipComplex));
	hipMemcpy(d_input, h_input, L*M*N * sizeof(hipComplex), hipMemcpyHostToDevice);
	//(hipComplex*)matrix;
	dim3 blockSize(1024, 1, 1);
	dim3 gridSize((L*N*N) / blockSize.x + 1, 1, 1);

	cout << "L,M,N:" << L << "," << M << "," << N << ",\n"
		<< "Input size " << L*M*N << " Out Size " << L*N*N
		<< endl;

	if (L*N*N > 1024) {
		doProblemKernel<<<gridSize, blockSize>>>(d_input, d_output, L, M, N);
	}
	else {
		doProblemKernel<<<1, L*N*N>>>(d_input, d_output, L, M, N);
	}
	err = hipGetLastError();
	hipComplex* h_output = (hipComplex*)malloc(L*N*N * sizeof(hipComplex));
	hipMemcpy(h_output, d_output, L*N*N * sizeof(hipComplex), hipMemcpyDeviceToHost);

	complex<float>* kerneloutput = (complex<float>*)h_output;


	for (int l = 0; l < L; l++) {
		for (int m = 0; m < N; m++) {
			for (int n = 0; n < N; n++) {
				//cout << matrix[l][m][n];
				cout << kerneloutput[l*N*N + m * N + n];
			}
			cout << endl;
		}
		cout << endl;
	}

	outfile.open("dat\\kernelsoutput.txt", 'w');
	for (int l = 0; l < L; l++) {
		for (int m = 0; m < N; m++) {
			for (int n = 0; n < N; n++) {
				//cout << matrix[l][m][n];
				outfile << kerneloutput[l*N*N + m * N + n];
			}
			outfile << endl;
		}
		outfile << endl;
	}
	outfile.close();
}

// Run program: Ctrl + F5 or Debug > Start Without Debugging menu
// Debug program: F5 or Debug > Start Debugging menu

// Tips for Getting Started: 
//   1. Use the Solution Explorer window to add/manage files
//   2. Use the Team Explorer window to connect to source control
//   3. Use the Output window to see build output and other messages
//   4. Use the Error List window to view errors
//   5. Go to Project > Add New Item to create new code files, or Project > Add Existing Item to add existing code files to the project
//   6. In the future, to open this project again, go to File > Open > Project and select the .sln file
